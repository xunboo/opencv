#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "math.hpp"
#include "array.hpp"
#include "limits.hpp"
#include "types.hpp"
#include "grid_stride_loop.hpp"
#include "execution.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/tensor.hpp"
#include "../cuda4dnn/csl/span.hpp"

#include <opencv2/core.hpp>

#include <cstddef>
#include <vector>

namespace cv { namespace dnn { namespace cuda4dnn { namespace csl  { namespace kernels {

    using index_type = gpu::index_type;
    using size_type = gpu::size_type;

    namespace raw {

        template <class T, std::size_t N>
        using array = utils::array<T, N>;

        template <class T, std::size_t Order>
        __global__ void max_pooling_with_indices(
            span<T> output, span<T> indices, view<T> input, size_type channels,
            array<size_type, Order> out_spatial_dims, array<size_type, Order> in_spatial_dims,
            array<size_type, Order> window_size, array<size_type, Order> strides, array<size_type, Order> padding_left)
        {
            /* every element in the output is mapped to a window in the input and each thread processes several windows */
            for (auto idx : grid_stride_range(output.size())) {
                size_type out_spatial_size = 1;
                array<index_type, Order> window_idx;
                for (int i = Order - 1; i >= 0; i--) {
                    window_idx[i] = (idx / out_spatial_size) % out_spatial_dims[i];
                    out_spatial_size *= out_spatial_dims[i];
                }

                const index_type n = idx / (out_spatial_size * channels);
                const index_type c = (idx / out_spatial_size) % channels;

                array<index_type, Order> start;
                for(int i = 0; i < Order; i++)
                    start[i] = window_idx[i] * strides[i] - padding_left[i];

                array<index_type, Order> end;
                for (int i = 0; i < Order; i++) {
                    using utils::min;
                    end[i] = min<index_type>(start[i] + window_size[i], in_spatial_dims[i]);
                }

                for (int i = 0; i < Order; i++) {
                    using utils::max;
                    start[i] = max(start[i], 0);
                }

                T max_value = gpu::numeric_limits<T>::lowest();
                index_type max_idx = -1;

                size_type in_spatial_size = 1;
                for (int i = 0; i < Order; i++)
                    in_spatial_size *= in_spatial_dims[i];

                const auto outer_offset =  (n * channels + c) * in_spatial_size;
                if (Order == 2) {
                    array<index_type, Order> idx;
                    for (idx[0] = start[0]; idx[0] != end[0]; idx[0]++) {
                        for (idx[1] = start[1]; idx[1] != end[1]; idx[1]++) {
                            index_type offset = 0;
                            index_type stride = 1;
                            for (int i = Order - 1; i >= 0; i--) {
                                offset += stride * idx[i];
                                stride *= in_spatial_dims[i];
                            }

                            if (input[outer_offset + offset] > max_value) {
                                max_idx = offset;
                                max_value = input[outer_offset + offset];
                            }
                        }
                    }
                } else if(Order == 3) {
                    array<index_type, Order> idx;
                    for (idx[0] = start[0]; idx[0] != end[0]; idx[0]++) {
                        for (idx[1] = start[1]; idx[1] != end[1]; idx[1]++) {
                            for (idx[2] = start[2]; idx[2] != end[2]; idx[2]++) {
                                index_type offset = 0;
                                index_type stride = 1;
                                for (int i = Order - 1; i >= 0; i--) {
                                    offset += stride * idx[i];
                                    stride *= in_spatial_dims[i];
                                }

                                if (input[outer_offset + offset] > max_value) {
                                    max_idx = offset;
                                    max_value = input[outer_offset + offset];
                                }
                            }
                        }
                    }
                }

                output[idx] = max_value;
                indices[idx] = max_idx;
            }
        }

        template <class T, std::size_t Order>
        __global__ void max_unpooling(
            span<T> output, view<T> input, view<T> indices, size_type channels,
            array<size_type, Order> out_spatial_dims, array<size_type, Order> in_spatial_dims,
            array<size_type, Order> window_size, array<size_type, Order> strides, array<size_type, Order> padding_left)
        {
            /* the output has already been zero filled */
            /* Every input value represents a window in the output. The max unpooling operation
             * copies the input value to exactly one location in the output window which is given
             * by the indices tensor.
             */
            for (auto idx : grid_stride_range(input.size())) {
                size_type in_spatial_size = 1;
                array<index_type, Order> window_idx;
                for (int i = Order - 1; i >= 0; i--) {
                    window_idx[i] = (idx / in_spatial_size) % in_spatial_dims[i];
                    in_spatial_size *= in_spatial_dims[i];
                }

                const index_type n = idx / (in_spatial_size * channels);
                const index_type c = (idx / in_spatial_size) % channels;

                array<index_type, Order> start;
                for (int i = 0; i < Order; i++) {
                    using utils::min;
                    using utils::max;
                    start[i] = max(0, min(window_idx[i] * strides[i] - padding_left[i], out_spatial_dims[i] - 1));
                }

                size_type out_spatial_size = 1;
                for (int i = 0; i < Order; i++)
                    out_spatial_size *= out_spatial_dims[i];

                index_type outer_offset = (n * channels + c) * out_spatial_size;
                output[outer_offset + static_cast<index_type>(indices[idx])] = input[idx];
            }
        }
    }

    template <class T, std::size_t Order> static
    void launch_max_pooling_kernel(
        const Stream& stream,
        span<T> output, span<T> indices, view<T> input, std::size_t channels,
        const std::vector<std::size_t>& out_spatial_dims, const std::vector<std::size_t>& in_spatial_dims,
        const std::vector<std::size_t>& window_size,
        const std::vector<std::size_t>& strides, const std::vector<std::size_t>& padding_left)
    {
        CV_Assert(out_spatial_dims.size() == Order);
        CV_Assert(in_spatial_dims.size() == Order);
        CV_Assert(window_size.size() == Order);
        CV_Assert(strides.size() == Order);
        CV_Assert(padding_left.size() == Order);
        CV_Assert(indices.size() == output.size());

        utils::array<size_type, Order> out_spatial_dims_k, in_spatial_dims_k;
        out_spatial_dims_k.assign(std::begin(out_spatial_dims), std::end(out_spatial_dims));
        in_spatial_dims_k.assign(std::begin(in_spatial_dims), std::end(in_spatial_dims));

        utils::array<size_type, Order> window_size_k, strides_k, padding_left_k;
        window_size_k.assign(std::begin(window_size), std::end(window_size));
        strides_k.assign(std::begin(strides), std::end(strides));
        padding_left_k.assign(std::begin(padding_left), std::end(padding_left));

        auto kernel = raw::max_pooling_with_indices<T, Order>;
        auto policy = make_policy(kernel, output.size(), 0, stream);
        launch_kernel(kernel, policy, output, indices, input, channels,
            out_spatial_dims_k, in_spatial_dims_k, window_size_k, strides_k, padding_left_k);
    }

    template <class T>
    void max_pooling_with_indices(
        const Stream& stream,
        TensorSpan<T> output, TensorSpan<T> indices, TensorView<T> input,
        const std::vector<std::size_t>& window_size, const std::vector<std::size_t>& strides,
        const std::vector<std::size_t>& padding_left)
    {
        auto order = window_size.size();
        CV_Assert(strides.size() == order);
        CV_Assert(padding_left.size() == order);
        CV_Assert(2 <= order && order <= 3);

        CV_Assert(is_shape_same(output, indices));
        CV_Assert(output.rank() == order + 2);
        CV_Assert(input.rank() == order + 2);

        CV_Assert(input.get_axis_size(1) == output.get_axis_size(1));

        std::vector<std::size_t> out_spatial_dims(order), in_spatial_dims(order);
        for (int i = 0; i < order; i++) {
            in_spatial_dims[i] = input.get_axis_size(2 + i);
            out_spatial_dims[i] = output.get_axis_size(2 + i);
        }

        std::size_t channels = input.get_axis_size(1);
        if (order == 3) {
            launch_max_pooling_kernel<T, 3>(stream, output, indices, input, channels,
                out_spatial_dims, in_spatial_dims, window_size, strides, padding_left);
        } else if (order == 2) {
            launch_max_pooling_kernel<T, 2>(stream, output, indices, input, channels,
                out_spatial_dims, in_spatial_dims, window_size, strides, padding_left);
        }
    }

    template void max_pooling_with_indices(const Stream&,
        TensorSpan<__half>, TensorSpan<__half>, TensorView<__half>,
        const std::vector<std::size_t>&, const std::vector<std::size_t>&,
        const std::vector<std::size_t>&);

    template void max_pooling_with_indices(const Stream&,
        TensorSpan<float>, TensorSpan<float>, TensorView<float>,
        const std::vector<std::size_t>&, const std::vector<std::size_t>&,
        const std::vector<std::size_t>&);

    template void max_pooling_with_indices(const Stream&,
        TensorSpan<double>, TensorSpan<double>, TensorView<double>,
        const std::vector<std::size_t>&, const std::vector<std::size_t>&,
        const std::vector<std::size_t>&);

    template <class T, std::size_t Order> static
    void launch_max_unpooling_kernel(
        const Stream& stream,
        span<T> output, view<T> input, view<T> indices, std::size_t channels,
        const std::vector<std::size_t>& out_spatial_dims, const std::vector<std::size_t>& in_spatial_dims,
        const std::vector<std::size_t>& window_size,
        const std::vector<std::size_t>& strides, const std::vector<std::size_t>& padding_left)
    {
        CV_Assert(out_spatial_dims.size() == Order);
        CV_Assert(in_spatial_dims.size() == Order);
        CV_Assert(window_size.size() == Order);
        CV_Assert(strides.size() == Order);
        CV_Assert(padding_left.size() == Order);
        CV_Assert(indices.size() == input.size());

        utils::array<size_type, Order> out_spatial_dims_k, in_spatial_dims_k;
        out_spatial_dims_k.assign(std::begin(out_spatial_dims), std::end(out_spatial_dims));
        in_spatial_dims_k.assign(std::begin(in_spatial_dims), std::end(in_spatial_dims));

        utils::array<size_type, Order> window_size_k, strides_k, padding_left_k;
        window_size_k.assign(std::begin(window_size), std::end(window_size));
        strides_k.assign(std::begin(strides), std::end(strides));
        padding_left_k.assign(std::begin(padding_left), std::end(padding_left));

        auto kernel = raw::max_unpooling<T, Order>;
        auto policy = make_policy(kernel, input.size(), 0, stream);
        launch_kernel(kernel, policy, output, input, indices, channels,
            out_spatial_dims_k, in_spatial_dims_k, window_size_k, strides_k, padding_left_k);
    }

    template <class T>
    void max_unpooling(
        const Stream& stream,
        TensorSpan<T> output, TensorView<T> input, TensorView<T> indices,
        const std::vector<std::size_t>& window_size, const std::vector<std::size_t>& strides,
        const std::vector<std::size_t>& padding_left)
    {
        auto order = window_size.size();
        CV_Assert(strides.size() == order);
        CV_Assert(padding_left.size() == order);
        CV_Assert(2 <= order && order <= 3);

        CV_Assert(is_shape_same(input, indices));
        CV_Assert(output.rank() == order + 2);
        CV_Assert(input.rank() == order + 2);

        CV_Assert(input.get_axis_size(1) == output.get_axis_size(1));

        std::vector<std::size_t> out_spatial_dims(order), in_spatial_dims(order);
        for (int i = 0; i < order; i++) {
            in_spatial_dims[i] = input.get_axis_size(2 + i);
            out_spatial_dims[i] = output.get_axis_size(2 + i);
        }

        std::size_t channels = input.get_axis_size(1);
        if (order == 3) {
            launch_max_unpooling_kernel<T, 3>(stream, output, input, indices, channels,
                out_spatial_dims, in_spatial_dims, window_size, strides, padding_left);
        } else if (order == 2) {
            launch_max_unpooling_kernel<T, 2>(stream, output, input, indices, channels,
                out_spatial_dims, in_spatial_dims, window_size, strides, padding_left);
        }
    }

    template void max_unpooling(const Stream&,
        TensorSpan<__half>, TensorView<__half>, TensorView<__half>,
        const std::vector<std::size_t>&, const std::vector<std::size_t>&,
        const std::vector<std::size_t>&);

    template void max_unpooling(const Stream&,
        TensorSpan<float>, TensorView<float>, TensorView<float>,
        const std::vector<std::size_t>&, const std::vector<std::size_t>&,
        const std::vector<std::size_t>&);

    template void max_unpooling(const Stream&,
        TensorSpan<double>, TensorView<double>, TensorView<double>,
        const std::vector<std::size_t>&, const std::vector<std::size_t>&,
        const std::vector<std::size_t>&);


}}}}} /*  cv::dnn::cuda4dnn::csl::kernels */
