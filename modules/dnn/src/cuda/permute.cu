#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "array.hpp"
#include "types.hpp"
#include "grid_stride_loop.hpp"
#include "execution.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/tensor.hpp"
#include "../cuda4dnn/csl/span.hpp"

#include <opencv2/core.hpp>

#include <cstddef>

namespace cv { namespace dnn { namespace cuda4dnn { namespace csl  { namespace kernels {

    using index_type = gpu::index_type;
    using size_type = gpu::size_type;

    namespace raw {
        template <class T, std::size_t N>
        using array = utils::array<T, N>;

        template <class T, std::size_t N>
        __global__ void permute(
            array<index_type, N> axis_order,
            span<T> output, array<size_type, N> outStrides,
            view<T> input, array<size_type, N> inStrides)
        {
            for (auto i : grid_stride_range(input.size())) {
                index_type oldPosition = 0;
                index_type newPosition = i;

                for (int j = 0; j < N; j++)
                {
                    auto order = axis_order[j];
                    oldPosition += (newPosition / outStrides[j]) * inStrides[order];
                    newPosition %= outStrides[j];
                }

                output[i] = input[oldPosition];
            }
        }
    }

    template <class T, std::size_t N> static
    void launch_permute_kernel(
        const Stream& stream,
        const std::vector<std::size_t>& order,
        span<T> output, const std::vector<std::size_t>& outStride,
        view<T> input, const std::vector<std::size_t>& inStride)
    {
        CV_Assert(order.size() == N);
        CV_Assert(outStride.size() == N);
        CV_Assert(inStride.size() == N);

        utils::array<index_type, N> order_k;
        order_k.assign(std::begin(order), std::end(order));

        utils::array<size_type, N> outStride_k, inStride_k;
        outStride_k.assign(std::begin(outStride), std::end(outStride));
        inStride_k.assign(std::begin(inStride), std::end(inStride));

        auto kernel = raw::permute<T, N>;
        auto policy = make_policy(kernel, input.size(), 0, stream);
        launch_kernel(kernel, policy, order_k, output, outStride_k, input, inStride_k);
    }

    template <class T>
    void permute(
        const Stream& stream,
        TensorSpan<T> output, TensorView<T> input,
        std::vector<std::size_t> order)
    {
        CV_Assert(output.rank() == input.rank());
        CV_Assert(input.rank() == order.size());
        CV_Assert(input.size() == output.size());

        /* if there are axes in the begining that can be squeezed out and are not permuted,
         * we can eliminate those axes
         *
         * Reasoning:
         * ----------
         * Suppose an item's indices in the input tensor be [i1, i2, ...]. The indices in the
         * output tensor will be some permutation of the input tensor indices. Let the output
         * tensor indices be [o1, o2, ...]. The permutation operation essentially copies items
         * from the input tensor to new locations in the output tensor as dictated by the indices.
         *
         * If the size of the first axis of the input and output tensor is one and these axes are
         * not involved in any permutation, i.e. order[0] = 0, the input and output indicies for
         * all the elements will be of the form be [0, i2, ...] and [0, o2, ...] respectively.
         * The first index does not contribute to the element's address calculation and hence does
         * nothing other than consuming few cycles.
         */
        while (order[0] == 0 && input.get_axis_size(0) == 1 && output.get_axis_size(0) == 1) {
            /* remove the axes */
            input.squeeze(0);
            output.squeeze(0);

            /* when we remove axis zero, the axis index will be one less than the previous index
             * for all the remaining axes
             */
            order.erase(order.begin());
            for (auto& axis : order)
                axis--;

            /* optimizations should not break the invariants for the permutation operation */
            CV_Assert(output.rank() == input.rank());
            CV_Assert(input.rank() == order.size());
            CV_Assert(input.size() == output.size());
        }

        auto rank = output.rank();
        auto inShape = input.shape_as_vector();
        auto outShape = output.shape_as_vector();

        std::vector<std::size_t> inStride(rank), outStride(rank);
        inStride.back() = 1;
        outStride.back() = 1;
        /* garbage, ..., garbage, 1 */

        std::copy(std::begin(inShape) + 1, std::end(inShape), std::begin(inStride));
        std::copy(std::begin(outShape) + 1, std::end(outShape), std::begin(outStride));
        /* dim[0], dim[1], ..., dim[-1], 1 */

        std::partial_sum(inStride.rbegin(), inStride.rend(), inStride.rbegin(), std::multiplies<std::size_t>());
        std::partial_sum(outStride.rbegin(), outStride.rend(), outStride.rbegin(), std::multiplies<std::size_t>());
        /* stride[0], stride[1], ..., stride[-2], 1 */

        CV_Assert(rank >= 2 && rank <= 5);
        if (rank == 5) {
            launch_permute_kernel<T, 5>(stream, order, output, outStride, input, inStride);
        } else if (rank == 4) {
            launch_permute_kernel<T, 4>(stream, order, output, outStride, input, inStride);
        } else if (rank == 3) {
            launch_permute_kernel<T, 3>(stream, order, output, outStride, input, inStride);
        } else if (rank == 2) {
            launch_permute_kernel<T, 2>(stream, order, output, outStride, input, inStride);
        }
    }

    template void permute(const Stream&, TensorSpan<__half>, TensorView<__half>, std::vector<std::size_t>);
    template void permute(const Stream&, TensorSpan<float>, TensorView<float>, std::vector<std::size_t>);
    template void permute(const Stream&, TensorSpan<double>, TensorView<double>, std::vector<std::size_t>);

}}}}} /*  cv::dnn::cuda4dnn::csl::kernels */
