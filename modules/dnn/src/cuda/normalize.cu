#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include "array.hpp"
#include "math.hpp"
#include "types.hpp"
#include "atomics.hpp"
#include "grid_stride_loop.hpp"
#include "execution.hpp"

#include "../cuda4dnn/csl/kernels.hpp"
#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/span.hpp"

#include <hip/hip_runtime.h>

#include <cstddef>

namespace cv { namespace dnn { namespace cuda4dnn { namespace csl  { namespace kernels {

    namespace raw {
        using index_type = gpu::index_type;
        using size_type = gpu::size_type;

        template <class T>
        __global__ void zero(span<T> output) {
            for (auto idx : grid_stride_range(output.size()))
                output[idx] = 0;
        }

        template <class T>
        __global__ void reduce_sum_abs(span<T> output, view<T> input, size_type outer_stride, size_type mid_stride) {
            for (auto idx : grid_stride_range(input.size())) {
                const index_type outer_idx = idx / outer_stride;
                const index_type inner_idx = idx % mid_stride;

                const index_type sum_idx = outer_idx * mid_stride + inner_idx;
                atomicAdd(&output[sum_idx], utils::abs(input[idx]));
            }
        }

        template <class T>
        __global__ void reciprocal(span<T> output, T epsilon) {
            for (auto idx : grid_stride_range(output.size()))
                output[idx] = 1 / (output[idx] + epsilon);
        }

        template <class T>
        __global__ void reduce_sum_squared(span<T> output, view<T> input, size_type outer_stride, size_type mid_stride) {
           for (auto idx : grid_stride_range(input.size())) {
                const index_type outer_idx = idx / outer_stride;
                const index_type inner_idx = idx % mid_stride;

                const index_type sum_idx = outer_idx * mid_stride + inner_idx;
                atomicAdd(&output[sum_idx], input[idx] * input[idx]);
           }
        }

        template <class T>
        __global__ void rsqrt(span<T> output, T epsilon) {
            for (auto idx : grid_stride_range(output.size()))
                output[idx] = utils::rsqrt(output[idx] + epsilon);
        }

        template <class T>
        __global__ void apply_norm(span<T> output, view<T> input, size_type outer_stride, size_type mid_stride, view<T> sums)
        {
            for (auto idx : grid_stride_range(output.size())) {
                const index_type outer_idx = idx / outer_stride;
                const index_type inner_idx = idx % mid_stride;

                const index_type sum_idx = outer_idx * mid_stride + inner_idx;
                output[idx] = input[idx] * sums[sum_idx];
            }
        }
    }

    template <class T>
    void normalize(
        const Stream& stream,
        span<T> output,
        view<T> input, std::size_t outer_size, std::size_t mid_size, std::size_t inner_size, T norm, T epsilon,
        span<T> workspace)
    {
        CV_Assert(norm == 1 || norm == 2);
        CV_Assert(workspace.size() >= outer_size * inner_size);

        auto sums = span<T>(workspace.data(), outer_size * inner_size);

        auto zero_kernel = raw::zero<T>;
        auto policy = make_policy(zero_kernel, sums.size(), 0, stream);
        launch_kernel(zero_kernel, policy, sums);

        if (norm == 1) {
            auto reduce_kernel = raw::reduce_sum_abs<T>;
            policy = make_policy(reduce_kernel, input.size(), 0, stream);
            launch_kernel(reduce_kernel, policy, sums, input, mid_size * inner_size, inner_size);

            auto reciprocal_kernel = raw::reciprocal<T>;
            policy = make_policy(reciprocal_kernel, sums.size(), 0, stream);
            launch_kernel(reciprocal_kernel, policy, sums, epsilon);
        } else {
            auto reduce_kernel = raw::reduce_sum_squared<T>;
            policy = make_policy(reduce_kernel, input.size(), 0, stream);
            launch_kernel(reduce_kernel, policy, sums, input, mid_size * inner_size, inner_size);

            auto rsqrt_kernel = raw::rsqrt<T>;
            policy = make_policy(rsqrt_kernel, sums.size(), 0, stream);
            launch_kernel(rsqrt_kernel, policy, sums, epsilon);
        }

        auto scale_kernel = raw::apply_norm<T>;
        policy = make_policy(scale_kernel, output.size(), 0, stream);
        launch_kernel(scale_kernel, policy, output, input, mid_size * inner_size, inner_size, sums);
    }

    template void normalize<float>(const Stream&, span<float>, view<float>, std::size_t, std::size_t, std::size_t, float, float, span<float>);
    template void normalize<double>(const Stream&, span<double>, view<double>, std::size_t, std::size_t, std::size_t, double, double, span<double>);

}}}}} /*  cv::dnn::cuda4dnn::csl::kernels */
