#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "array.hpp"
#include "math.hpp"
#include "types.hpp"
#include "atomics.hpp"
#include "grid_stride_range.hpp"
#include "execution.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/span.hpp"

#include "../cuda4dnn/kernels/fill.hpp"
#include "../cuda4dnn/kernels/scale_shift.hpp"

#include <opencv2/core.hpp>

#include <cstddef>

using namespace cv::dnn::cuda4dnn::csl;
using namespace cv::dnn::cuda4dnn::csl::device;

namespace cv { namespace dnn { namespace cuda4dnn { namespace kernels {

    namespace raw {
        template <class T>
        __global__ void reduce_sum_abs(span<T> output, view<T> input, size_type outer_stride, size_type mid_stride) {
            for (auto idx : grid_stride_range(input.size())) {
                const index_type outer_idx = idx / outer_stride;
                const index_type inner_idx = idx % mid_stride;

                const index_type sum_idx = outer_idx * mid_stride + inner_idx;
                atomicAdd(&output[sum_idx], device::abs(input[idx]));
            }
        }

        template <class T>
        __global__ void reciprocal(span<T> output, T epsilon) {
            for (auto idx : grid_stride_range(output.size()))
                output[idx] = T(1) / (output[idx] + epsilon);
        }

        template <class T>
        __global__ void reduce_sum_squared(span<T> output, view<T> input, size_type outer_stride, size_type mid_stride) {
           for (auto idx : grid_stride_range(input.size())) {
                const index_type outer_idx = idx / outer_stride;
                const index_type inner_idx = idx % mid_stride;

                const index_type sum_idx = outer_idx * mid_stride + inner_idx;
                atomicAdd(&output[sum_idx], input[idx] * input[idx]);
           }
        }

        template <class T>
        __global__ void rsqrt(span<T> output, T epsilon) {
            for (auto idx : grid_stride_range(output.size())) {
                using device::sqrt;
                output[idx] = T(1) / sqrt(output[idx] + epsilon);
            }
        }

        template <class T>
        __global__ void apply_norm(span<T> output, view<T> input, size_type outer_stride, size_type mid_stride, view<T> sums)
        {
            for (auto idx : grid_stride_range(output.size())) {
                const index_type outer_idx = idx / outer_stride;
                const index_type inner_idx = idx % mid_stride;

                const index_type sum_idx = outer_idx * mid_stride + inner_idx;
                output[idx] = input[idx] * sums[sum_idx];
            }
        }
    }

    template <class T>
    void normalize(
        const Stream& stream,
        span<T> output,
        view<T> input, std::size_t outer_size, std::size_t mid_size, std::size_t inner_size, std::size_t norm, T epsilon,
        span<T> workspace)
    {
        CV_Assert(output.size() == input.size());
        CV_Assert(output.size() == outer_size * mid_size * inner_size);
        CV_Assert(norm == 1 || norm == 2);
        CV_Assert(workspace.size() >= outer_size * inner_size);

        auto sums = span<T>(workspace.data(), outer_size * inner_size);

        fill<T>(stream, sums, 0.0);

        if (norm == 1) {
            auto reduce_kernel = raw::reduce_sum_abs<T>;
            auto policy = make_policy(reduce_kernel, input.size(), 0, stream);
            launch_kernel(reduce_kernel, policy, sums, input, mid_size * inner_size, inner_size);

            auto reciprocal_kernel = raw::reciprocal<T>;
            policy = make_policy(reciprocal_kernel, sums.size(), 0, stream);
            launch_kernel(reciprocal_kernel, policy, sums, epsilon);
        } else {
            auto reduce_kernel = raw::reduce_sum_squared<T>;
            auto policy = make_policy(reduce_kernel, input.size(), 0, stream);
            launch_kernel(reduce_kernel, policy, sums, input, mid_size * inner_size, inner_size);

            auto rsqrt_kernel = raw::rsqrt<T>;
            policy = make_policy(rsqrt_kernel, sums.size(), 0, stream);
            launch_kernel(rsqrt_kernel, policy, sums, epsilon);
        }

        auto scale_kernel = raw::apply_norm<T>;
        auto policy = make_policy(scale_kernel, output.size(), 0, stream);
        launch_kernel(scale_kernel, policy, output, input, mid_size * inner_size, inner_size, sums);
    }

    template void normalize(const Stream&, span<__half>, view<__half>, std::size_t, std::size_t, std::size_t, std::size_t, __half, span<__half>);
    template void normalize(const Stream&, span<float>, view<float>, std::size_t, std::size_t, std::size_t, std::size_t, float, span<float>);
    template void normalize(const Stream&, span<double>, view<double>, std::size_t, std::size_t, std::size_t, std::size_t, double, span<double>);

}}}} /* cv::dnn::cuda4dnn::kernels */
